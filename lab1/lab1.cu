#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<len) {
    out[i] = in1[i] + in2[i];
  }

}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;

  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  //@@ Allocate GPU memory here
  //device memory
  int size = inputLength*sizeof(float);
  float *input1, *input2, *output;
  hipMalloc((void **) &input1, size);
  hipMalloc((void **) &input2, size);
  hipMalloc((void **) &output, size);

  //@@ Copy memory to the GPU here
  hipMemcpy(input1, hostInput1, size, hipMemcpyHostToDevice);
  hipMemcpy(input2, hostInput2, size, hipMemcpyHostToDevice);
  hipMemcpy(output, hostOutput, size, hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(inputLength/256.0),1,1);
  dim3 DimBlock(256,1,1);

  //@@ Launch the GPU Kernel here to perform CUDA computation
  vecAdd<<<DimGrid, DimBlock>>>(input1, input2, output, inputLength);

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, output, size, hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(input1); hipFree(input2); hipFree(output);

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
