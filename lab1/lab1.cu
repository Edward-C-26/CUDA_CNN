#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i<len) {
    out[i] = in1[i] + in2[i];
  }

}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;

  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  //@@ Allocate GPU memory here
  //device memory
  float *input1, *input2, *output;
  hipMalloc((void **) &input1, inputLength);
  hipMalloc((void **) &input2, inputLength);
  hipMalloc((void **) &output, inputLength);

  //@@ Copy memory to the GPU here
  hipMemcpy(input1, hostInput1, inputLength, hipMemcpyHostToDevice);
  hipMemcpy(input2, hostInput2, inputLength, hipMemcpyHostToDevice);
  hipMemcpy(output, hostOutput, inputLength, hipMemcpyHostToDevice);

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(ceil(inputLength/256.0),1,1);
  dim3 DimBlock(256,1,1);

  //@@ Launch the GPU Kernel here to perform CUDA computation
  vecAdd<<<DimGrid, DimBlock>>>(input1, input2, output, inputLength);

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, output, inputLength, hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(hostInput1); hipFree(hostInput2); hipFree(hostOutput);

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
