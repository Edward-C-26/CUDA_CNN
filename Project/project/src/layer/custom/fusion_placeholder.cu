#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256

/*
Test batch size: 10000
Loading fashion-mnist data...Done
Loading model...Done
Conv-GPU==
Layer Time: 211.331 ms
Op Time: 50.5587 ms
Conv-GPU==
Layer Time: 152.88 ms
Op Time: 30.3197 ms

Test Accuracy: 0.8714
*/

__global__ void matrix_unrolling_kernel(const float *input, const float *weight_matrix, float *C,
                                        const int Batch, const int Channel,
                                        const int Height, const int Width,
                                        const int K, int Map_out, int Height_unrolled, int Width_unrolled) {
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define out_4d(i3, i2, i1, i0) C[(size_t)(i3 * (Map_out * Height * Width)) + (size_t)((i2) * (Height_out * Width_out)) + (size_t)((i1) * (Width_out)) + (size_t)(i0)]
    
    // Matmul
    __shared__ float tileA[TILE_WIDTH * TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH * TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    int out_size = Height_out * Width_out;

    float val = 0;

    int batch = col / out_size;
    int height_fuse = (col % out_size) /  Width_out;
    int width_fuse = (col % out_size) % Width_out;

    for (int tileId = 0; tileId < (Height_unrolled - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < Map_out && tileId * TILE_WIDTH + tx < Height_unrolled) {
            tileA[ty*16 + tx] = weight_matrix[row * Height_unrolled + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty*16 + tx] = 0;
        }
        if (col < Width_unrolled && tileId * TILE_WIDTH + ty < Height_unrolled) {
            int channel = (tileId * TILE_WIDTH + ty) / K / K;
            int row_offset = (tileId * TILE_WIDTH + ty) % (K*K) / K;
            int col_offset = (tileId * TILE_WIDTH + ty) % (K*K) % K;

            tileB[ty*16 +tx] = in_4d(batch, channel, (height_fuse + row_offset), (width_fuse + col_offset));
            
        } else {
            tileB[ty*16 +tx] = 0;
        }
        __syncthreads();

        if (row < Height_unrolled && col < Width_unrolled) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty*16+i] * tileB[i*16+tx];
            }
        }
        __syncthreads();
    }

    // Permute occurs here
    if (row < Height_unrolled && col < Width_unrolled) {
        out_4d(batch, row, height_fuse, width_fuse) = val; 
    }

}


__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    size_t i_size = Batch * Channel * Height * Width * sizeof(float);
    size_t m_size = Map_out * Channel * K * K * sizeof(float);
    size_t o_size = Batch * Map_out * Height_out * Width_out * sizeof(float);

    hipMalloc((void**) device_input_ptr, i_size);
    hipMalloc((void**) device_mask_ptr, m_size);
    hipMalloc((void**) device_output_ptr, o_size);

    hipMemcpy(*device_input_ptr, host_input, i_size, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, m_size, hipMemcpyHostToDevice);

    // Error Check
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Height_unrolled = Channel * K * K;
    const int Width_unrolled = Batch * Height_out * Width_out;


    // TODO: Set the kernel dimensions and call the matrix unrolling kernel.
    int width_grid = Channel > Map_out ? Channel: Map_out;
    dim3 block(TILE_WIDTH, TILE_WIDTH, 1);
    dim3 grid(ceil(1.0f * Width_unrolled / TILE_WIDTH), ceil(1.0f * width_grid / TILE_WIDTH), 1);
    matrix_unrolling_kernel<<<grid, block>>>(device_input, device_mask, device_output, Batch, Channel, Height, Width, K, Map_out, Height_unrolled, Width_unrolled);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    hipMemcpy(host_output, device_output, Batch * Map_out * (Height - K + 1) * (Width - K + 1) * sizeof(float), hipMemcpyDeviceToHost);

    // TODO: Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}