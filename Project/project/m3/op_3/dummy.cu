#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16
#define BLOCK_SIZE 256

/* NOTE: A bit too much work, saving this at the end
op 3: Sweeping various parameters to find best values (block sizes, amount of thread coarsening) -- requires tables/graphs in Report
code: NOT COMPELTED
profiling: NOT COMPELTED



*/

__global__ void matrix_unrolling_kernel(const float* __restrict__ input, float* __restrict__ output,
                                        const int Batch, const int  Channel,
                                        const int  Height, const int  Width,
                                        const int  K) {
    /*
    Modify this function to implement the input matrix unrolling kernel.

    Function paramter definitions:
    input - input
    output - output
    Batch - batch_size (number of images in x)
    Channel - number of input feature maps
    Height - input height dimension
    Width - input width dimension
    K - kernel height and width (K x K)
    */
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    // (void)Height_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)Width_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)

    #define in_4d(i3, i2, i1, i0) input[(i3) * (Channel * Height * Width) + (i2) * (Height * Width) + (i1) * (Width) + i0]
    #define out_3d(i2, i1, i0) output[(size_t)((i2) * (Height_out * Batch * Width_out)) + (size_t)((i1) * (Height_out * Width_out)) + (size_t)(i0)]

    // TODO: Insert your input matrix unrolling kernel code here
    int t = blockIdx.x * blockDim.x + threadIdx.x;
    int out_size = Height_out * Width_out;

    if (t < Batch * out_size) {
        size_t b = t / out_size;                  // Batch index
        size_t h_out = (t / Width_out) % Height_out;
        size_t w_out = t % Width_out;

        for (int c = 0; c < Channel; c++) {
            int w_base = c * (K * K);             // Base offset for each channel

            for (int p = 0; p < K; p++) {
                for (int q = 0; q < K; q++) {
                    size_t h_unroll = w_base + p * K + q;
                    size_t w_unroll = h_out * Width_out + w_out;

                    // Ensure out_3d and in_4d macros align correctly with indexing
                    out_3d(h_unroll, b, w_unroll) = in_4d(b, c, h_out + p, w_out + q);
                }
            }
        }
    }
    #undef out_3d
    #undef in_4d
}

// Tiled matrix multiplication kernel. Computes C = AB
// You don't need to modify this kernel.
__global__ void matrixMultiplyShared(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
                                     int  numARows, int  numAColumns,
                                     int  numBRows, int  numBColumns,
                                     int  numCRows, int  numCColumns)
{
    __shared__ float tileA[TILE_WIDTH][TILE_WIDTH];
    __shared__ float tileB[TILE_WIDTH][TILE_WIDTH];

    int by = blockIdx.y, bx = blockIdx.x, ty = threadIdx.y, tx = threadIdx.x;

    int row = by * TILE_WIDTH + ty, col = bx * TILE_WIDTH + tx;
    float val = 0;

    for (int tileId = 0; tileId < (numAColumns - 1) / TILE_WIDTH + 1; tileId++) {
        if (row < numARows && tileId * TILE_WIDTH + tx < numAColumns) {
            tileA[ty][tx] = A[(size_t) row * numAColumns + tileId * TILE_WIDTH + tx];
        } else {
            tileA[ty][tx] = 0;
        }
        if (col < numBColumns && tileId * TILE_WIDTH + ty < numBRows) {
            tileB[ty][tx] = B[((size_t) tileId * TILE_WIDTH + ty) * numBColumns + col];
        } else {
            tileB[ty][tx] = 0;
        }
        __syncthreads();

        if (row < numCRows && col < numCColumns) {
            for (int i = 0; i < TILE_WIDTH; i++) {
                val += tileA[ty][i] * tileB[i][tx];
            }
        }
        __syncthreads();
    }

    if (row < numCRows && col < numCColumns) {
        C[row * numCColumns + col] = val;
    }
}

// Permutes the matmul result.
// The output feature map after matmul is of shape Map_out x Batch x Height_out x Width_out,
// and we need to permute it into Batch x Map_out x Height_out x Width_out.
// You don't need to modify this kernel.
__global__ void matrix_permute_kernel(const float* __restrict__ input, float* __restrict__ output, int  Map_out,
                                      int  Batch, int  image_size) {
    int b = blockIdx.y;
    int x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    if (x < image_size) {
        for (int m = 0; m < Map_out; m++) {
            output[b * Map_out * image_size + m * image_size + x] =
                    input[m * Batch * image_size + b * image_size + x];
        }
    }
}

//even thougth the host variables are access at most once here, making it "read only" could help
__host__ void GPUInterface::conv_forward_gpu_prolog(const float* __restrict__ host_output, const float* __restrict__ host_input, const float* __restrict__ host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Allocate memory and copy over the relevant data structures to the GPU
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;

    size_t i_size = Batch * Channel * Height * Width * sizeof(float);
    size_t m_size = Map_out * Channel * K * K * sizeof(float);
    size_t o_size = Batch * Map_out * Height_out * Width_out * sizeof(float);

    hipMalloc((void**) device_input_ptr, i_size);
    hipMalloc((void**) device_mask_ptr, m_size);
    hipMalloc((void**) device_output_ptr, o_size);

    hipMemcpy(*device_input_ptr, host_input, i_size, hipMemcpyHostToDevice);
    hipMemcpy(*device_mask_ptr, host_mask, m_size, hipMemcpyHostToDevice);

    // Error Check
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::conv_forward_gpu(float*  device_output, const float*  device_input, const float*  device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    const int Height_out = Height - K + 1;
    const int Width_out = Width - K + 1;
    const int Height_unrolled = Channel * K * K;
    const int Width_unrolled = Batch * Height_out * Width_out;

    float *unrolled_matrix;  // Pointer to device memory for storing the unrolled matrix
    float *matmul_output;    // Pointer to device memory for storing the result of matrix multiplication
    hipMalloc((void**)&unrolled_matrix, (size_t) Batch * Channel * K * K * Height_out * Width_out * sizeof(float));
    hipMalloc((void**)&matmul_output, Batch * Map_out * Height_out * Width_out * sizeof(float));

    // TODO: Set the kernel dimensions and call the matrix unrolling kernel.
    int num_threads = Batch * Height_out * Width_out * Channel;
    int num_blocks = ceil(1.0f * num_threads / BLOCK_SIZE);
    matrix_unrolling_kernel<<<num_blocks, BLOCK_SIZE>>>(device_input, unrolled_matrix, Batch, Channel, Height, Width, K);

    // TODO: Set the kernel dimensions and call the matmul kernel
    int matmul_w = ceil(1.0f * Width_unrolled / TILE_WIDTH); 
    int matmul_h = ceil(1.0f * Map_out / TILE_WIDTH);
    dim3 dimGrid(matmul_w, matmul_h, 1);
    dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
    matrixMultiplyShared<<<dimGrid, dimBlock>>>(device_mask, unrolled_matrix, matmul_output, Map_out, Height_unrolled, Height_unrolled, Width_unrolled, Map_out, Width_unrolled);

    // Permute the result of matrix multiplication
    const int out_image_size = Height_out * Width_out;
    dim3 permute_kernel_grid_dim((out_image_size - 1) / BLOCK_SIZE + 1, Batch, 1);
    matrix_permute_kernel<<<permute_kernel_grid_dim, BLOCK_SIZE>>>(
        matmul_output, device_output, Map_out, Batch, out_image_size
    );

    hipFree(matmul_output);
    hipFree(unrolled_matrix);
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float*  device_input, float*  device_mask, const int Batch, const int Map_out, const int Channel, const int Height, const int Width, const int K)
{
    // TODO: Copy the output back to host
    hipMemcpy(host_output, device_output, Batch * Map_out * (Height - K + 1) * (Width - K + 1) * sizeof(float), hipMemcpyDeviceToHost);

    // TODO: Free device memory
    hipFree(device_input);
    hipFree(device_output);
    hipFree(device_mask);
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}